// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify=expected,onhost %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify=expected,ondevice %s


// Tests to ensure that functions with host and device overloads in that are
// called outside of function bodies and variable initializers, e.g., in
// template arguments are resolved with respect to the declaration to which they
// belong.

// Opaque types used for tests:
struct DeviceTy {};
struct HostTy {};
struct HostDeviceTy {};
struct TemplateTy {};

struct TrueTy { static const bool value = true; };
struct FalseTy { static const bool value = false; };

// Select one of two types based on a boolean condition.
template <bool COND, typename T, typename F> struct select_type {};
template <typename T, typename F> struct select_type<true, T, F> { typedef T type; };
template <typename T, typename F> struct select_type<false, T, F> { typedef F type; };

template <bool C> struct check : public select_type<C, TrueTy, FalseTy> { };

// Check if two types are the same.
template<class T, class U> struct is_same : public FalseTy { };
template<class T> struct is_same<T, T> : public TrueTy { };

// A static assertion that fails at compile time if the expression E does not
// have type T.
#define ASSERT_HAS_TYPE(E, T) static_assert(is_same<decltype(E), T>::value);


// is_on_device() is true when called in a device context and false if called in a host context.
__attribute__((host)) constexpr bool is_on_device(void) { return false; }
__attribute__((device)) constexpr bool is_on_device(void) { return true; }


// this type depends on whether it occurs in host or device code
#define targetdep_t select_type<is_on_device(), DeviceTy, HostTy>::type

// Defines and typedefs with different values in host and device compilation.
#ifdef __CUDA_ARCH__
#define CurrentTarget DEVICE
typedef DeviceTy CurrentTargetTy;
typedef DeviceTy TemplateIfHostTy;
#else
#define CurrentTarget HOST
typedef HostTy CurrentTargetTy;
typedef TemplateTy TemplateIfHostTy;
#endif



// targetdep_t in function declarations should depend on the target of the
// declared function.
__attribute__((device)) targetdep_t decl_ret_early_device(void);
ASSERT_HAS_TYPE(decl_ret_early_device(), DeviceTy)

__attribute__((host)) targetdep_t decl_ret_early_host(void);
ASSERT_HAS_TYPE(decl_ret_early_host(), HostTy)

__attribute__((host,device)) targetdep_t decl_ret_early_host_device(void);
ASSERT_HAS_TYPE(decl_ret_early_host_device(), CurrentTargetTy)

// If the function target is specified too late and can therefore not be
// considered for overload resolution in targetdep_t, warn.
targetdep_t __attribute__((device)) decl_ret_late_device(void); // expected-warning {{target attribute has been ignored for overload resolution}}
ASSERT_HAS_TYPE(decl_ret_late_device(), HostTy)

// No warning necessary if the ignored attribute doesn't change the result.
targetdep_t __attribute__((host)) decl_ret_late_host(void);
ASSERT_HAS_TYPE(decl_ret_late_host(), HostTy)

targetdep_t __attribute__((host,device)) decl_ret_late_host_device(void); // expected-warning {{target attribute has been ignored for overload resolution}}
ASSERT_HAS_TYPE(decl_ret_late_host_device(), HostTy)

// An odd way of writing this, but it's possible.
__attribute__((device)) targetdep_t __attribute__((host)) decl_ret_early_device_late_host(void); // expected-warning {{target attribute has been ignored for overload resolution}}
ASSERT_HAS_TYPE(decl_ret_early_device_late_host(), DeviceTy)


// The same for function definitions and parameter types:
__attribute__((device)) targetdep_t ret_early_device(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

__attribute__((host)) targetdep_t ret_early_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

__attribute__((host, device)) targetdep_t ret_early_hostdevice(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_early_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

// The parameter is still after the attribute, so it needs no warning.
targetdep_t __attribute__((device)) // expected-warning {{target attribute has been ignored for overload resolution}}
ret_late_device(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_late_device({}), HostTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

targetdep_t __attribute__((host, device)) // expected-warning {{target attribute has been ignored for overload resolution}}
ret_late_hostdevice(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_late_hostdevice({}), HostTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

targetdep_t __attribute__((host)) ret_late_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

__attribute__((device)) targetdep_t __attribute__((host)) // expected-warning {{target attribute has been ignored for overload resolution}}
ret_early_device_late_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ret_early_device_late_host({}), DeviceTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

// The attribute is even later, so we can't choose the expected overload.
targetdep_t ret_verylate_device(targetdep_t x) __attribute__((device)) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ret_verylate_device({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

// It's possible to get two different wrong types:
targetdep_t __attribute__((device)) // expected-warning {{target attribute has been ignored for overload resolution}}
ret_late_device_verylate_host(targetdep_t x) __attribute__((host)) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ret_late_device_verylate_host({}), HostTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}


// Increasingly unusual ways to specify a return type:

// The attribute is specified much earlier than the overload happens, works as
// expected.
__attribute__((device)) auto autoret_early_device(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(autoret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

// The attribute is specified much earlier than the overload happens, works as
// expected.
__attribute__((host)) auto autoret_early_host(targetdep_t x) -> targetdep_t  {
  ASSERT_HAS_TYPE(autoret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

// The attribute is specified much earlier than the overload happens, works as
// expected.
__attribute__((host,device)) auto autoret_early_hostdevice(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(autoret_early_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}


// The attribute is still specified earlier than the overload happens, works as
// expected.
auto __attribute__((device)) autoret_late_device(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(autoret_late_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

// The attribute is still specified earlier than the overload happens, works as
// expected.
auto __attribute__((host)) autoret_late_host(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(autoret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

// The attribute is still specified earlier than the overload happens, works as
// expected.
auto __attribute__((host,device)) autoret_late_hostdevice(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(autoret_late_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}


// There should be no problem if the return type is inferred from an expression in the body:
auto __attribute__((device)) fullauto_device(targetdep_t x) {
  ASSERT_HAS_TYPE(x, DeviceTy)
  return (targetdep_t)(x);
}
ASSERT_HAS_TYPE(fullauto_device({}), DeviceTy)

auto __attribute__((host)) fullauto_host(targetdep_t x) {
  ASSERT_HAS_TYPE(x, HostTy)
  return (targetdep_t)(x);
}
ASSERT_HAS_TYPE(fullauto_host({}), HostTy)

// The return type is as expected, but the argument type precedes the attribute,
// so we don't get the right type for it.
auto fullauto_verylate_device(targetdep_t x) __attribute__((device)) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(x, HostTy)
  return targetdep_t();
}
ASSERT_HAS_TYPE(fullauto_verylate_device({}), DeviceTy)

auto fullauto_verylate_host(targetdep_t x) __attribute__((host)) {
  ASSERT_HAS_TYPE(x, HostTy)
  return targetdep_t();
}
ASSERT_HAS_TYPE(fullauto_verylate_host({}), HostTy)


// MS __declspec syntax:
__declspec(__device__) targetdep_t ms_ret_early_device(targetdep_t x) {
  ASSERT_HAS_TYPE(ms_ret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

__declspec(__host__) targetdep_t ms_ret_early_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ms_ret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

__declspec(__host__) __declspec(__device__) targetdep_t ms_ret_early_hostdevice(targetdep_t x) {
  ASSERT_HAS_TYPE(ms_ret_early_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

targetdep_t __declspec(__device__) ms_ret_late_device(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ms_ret_late_device({}), HostTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

targetdep_t __declspec(__host__) ms_ret_late_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ms_ret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

targetdep_t __declspec(__host__) __declspec(__device__) ms_ret_late_hostdevice(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ms_ret_late_hostdevice({}), HostTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

__declspec(__device__) targetdep_t __declspec(__host__) ms_ret_early_device_late_host(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ms_ret_early_device_late_host({}), DeviceTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

__declspec(__device__) auto ms_autoret_early_device(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(ms_autoret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

__declspec(__host__) auto ms_autoret_early_host(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(ms_autoret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

__declspec(__host__) __declspec(__device__) auto ms_autoret_early_hostdevice(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(ms_autoret_early_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}


auto __declspec(__device__) ms_autoret_late_device(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(ms_autoret_late_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

auto __declspec(__host__) ms_autoret_late_host(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(ms_autoret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

auto __declspec(__host__) __declspec(__device__) ms_autoret_late_hostdevice(targetdep_t x) -> targetdep_t {
  ASSERT_HAS_TYPE(ms_autoret_late_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}


// Class/Struct member functions:

struct MethodTests {
  __attribute__((device)) targetdep_t ret_early_device(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_early_device({}), DeviceTy)
    ASSERT_HAS_TYPE(x, DeviceTy)
    return {};
  }

  __attribute__((host)) targetdep_t ret_early_host(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_early_host({}), HostTy)
    ASSERT_HAS_TYPE(x, HostTy)
    return {};
  }

  __attribute__((host,device)) targetdep_t ret_early_hostdevice(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_early_hostdevice({}), CurrentTargetTy)
    ASSERT_HAS_TYPE(x, CurrentTargetTy)
    return {};
  }

  __attribute__((device)) auto autoret_early_device(targetdep_t x) -> targetdep_t {
    ASSERT_HAS_TYPE(autoret_early_device({}), DeviceTy)
    ASSERT_HAS_TYPE(x, DeviceTy)
    return {};
  }
  __attribute__((host)) auto autoret_early_host(targetdep_t x) -> targetdep_t {
    ASSERT_HAS_TYPE(autoret_early_host({}), HostTy)
    ASSERT_HAS_TYPE(x, HostTy)
    return {};
  }

  __attribute__((host,device)) auto autoret_early_hostdevice(targetdep_t x) -> targetdep_t {
    ASSERT_HAS_TYPE(autoret_early_hostdevice({}), CurrentTargetTy)
    ASSERT_HAS_TYPE(x, CurrentTargetTy)
    return {};
  }


  // Overloaded call happens in return type, attribute is after that.
  targetdep_t __attribute__((device)) ret_late_device(targetdep_t x) {  // expected-warning {{target attribute has been ignored for overload resolution}}
    ASSERT_HAS_TYPE(ret_late_device({}), HostTy)
    ASSERT_HAS_TYPE(x, DeviceTy)
    return {};
  }

  targetdep_t __attribute__((host)) ret_late_host(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_late_host({}), HostTy)
    ASSERT_HAS_TYPE(x, HostTy)
    return {};
  }

  targetdep_t __attribute__((host,device)) ret_late_hostdevice(targetdep_t x) {  // expected-warning {{target attribute has been ignored for overload resolution}}
    ASSERT_HAS_TYPE(ret_late_hostdevice({}), HostTy)
    ASSERT_HAS_TYPE(x, CurrentTargetTy)
    return {};
  }


  // Member declarations (tested in the 'tests' function further below):
  __attribute__((device)) targetdep_t decl_ret_early_device(void);
  __attribute__((host)) targetdep_t decl_ret_early_host(void);
  __attribute__((host,device)) targetdep_t decl_ret_early_hostdevice(void);
  targetdep_t __attribute__((device)) decl_ret_late_device(void);  // expected-warning {{target attribute has been ignored for overload resolution}}
  targetdep_t __attribute__((host)) decl_ret_late_host(void);
  targetdep_t __attribute__((host,device)) decl_ret_late_hostdevice(void);  // expected-warning {{target attribute has been ignored for overload resolution}}

  // for out of line definitions:
  __attribute__((device)) targetdep_t ool_ret_early_device(targetdep_t x);
  __attribute__((host)) targetdep_t ool_ret_early_host(targetdep_t x);
  __attribute__((host,device)) targetdep_t ool_ret_early_hostdevice(targetdep_t x);
  targetdep_t __attribute__((device)) ool_ret_late_device(targetdep_t x);  // expected-warning {{target attribute has been ignored for overload resolution}}
  targetdep_t __attribute__((host)) ool_ret_late_host(targetdep_t x);
  targetdep_t __attribute__((host,device)) ool_ret_late_hostdevice(targetdep_t x);  // expected-warning {{target attribute has been ignored for overload resolution}}

};

__attribute__((device)) targetdep_t MethodTests::ool_ret_early_device(targetdep_t x) {
  ASSERT_HAS_TYPE(ool_ret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

__attribute__((host)) targetdep_t MethodTests::ool_ret_early_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ool_ret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

__attribute__((host,device)) targetdep_t MethodTests::ool_ret_early_hostdevice(targetdep_t x) {
  ASSERT_HAS_TYPE(ool_ret_early_hostdevice({}), CurrentTargetTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}

targetdep_t __attribute__((device)) MethodTests::ool_ret_late_device(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ool_ret_late_device({}), HostTy)
  ASSERT_HAS_TYPE(x, DeviceTy)
  return {};
}

targetdep_t __attribute__((host)) MethodTests::ool_ret_late_host(targetdep_t x) {
  ASSERT_HAS_TYPE(ool_ret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(x, HostTy)
  return {};
}

targetdep_t __attribute__((host,device)) MethodTests::ool_ret_late_hostdevice(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
  ASSERT_HAS_TYPE(ool_ret_late_hostdevice({}), HostTy)
  ASSERT_HAS_TYPE(x, CurrentTargetTy)
  return {};
}


// members of templated structs should also work.
template <unsigned int N>
struct TemplateMethodTests {
  __attribute__((device)) targetdep_t ret_early_device(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_early_device({}), DeviceTy)
    ASSERT_HAS_TYPE(x, DeviceTy)
    return {};
  }

  __attribute__((host)) targetdep_t ret_early_host(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_early_host({}), HostTy)
    ASSERT_HAS_TYPE(x, HostTy)
    return {};
  }

  __attribute__((host,device)) targetdep_t ret_early_hostdevice(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_early_hostdevice({}), CurrentTargetTy)
    ASSERT_HAS_TYPE(x, CurrentTargetTy)
    return {};
  }

  __attribute__((device)) auto autoret_early_device(targetdep_t x) -> targetdep_t {
    ASSERT_HAS_TYPE(autoret_early_device({}), DeviceTy)
    ASSERT_HAS_TYPE(x, DeviceTy)
    return {};
  }

  __attribute__((host)) auto autoret_early_host(targetdep_t x) -> targetdep_t {
    ASSERT_HAS_TYPE(autoret_early_host({}), HostTy)
    ASSERT_HAS_TYPE(x, HostTy)
    return {};
  }

  __attribute__((host,device)) auto autoret_early_hostdevice(targetdep_t x) -> targetdep_t {
    ASSERT_HAS_TYPE(autoret_early_hostdevice({}), CurrentTargetTy)
    ASSERT_HAS_TYPE(x, CurrentTargetTy)
    return {};
  }

  targetdep_t __attribute__((device)) ret_late_device(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
    ASSERT_HAS_TYPE(ret_late_device({}), HostTy)
    ASSERT_HAS_TYPE(x, DeviceTy)
    return {};
  }

  targetdep_t __attribute__((host)) ret_late_host(targetdep_t x) {
    ASSERT_HAS_TYPE(ret_late_host({}), HostTy)
    ASSERT_HAS_TYPE(x, HostTy)
    return {};
  }

  targetdep_t __attribute__((host,device)) ret_late_hostdevice(targetdep_t x) { // expected-warning {{target attribute has been ignored for overload resolution}}
    ASSERT_HAS_TYPE(ret_late_hostdevice({}), HostTy)
    ASSERT_HAS_TYPE(x, CurrentTargetTy)
    return {};
  }


  __attribute__((device)) targetdep_t decl_ret_early_device(void);
  __attribute__((host)) targetdep_t decl_ret_early_host(void);
  __attribute__((host,device)) targetdep_t decl_ret_early_hostdevice(void);

  targetdep_t __attribute__((device)) decl_ret_late_device(void); // expected-warning {{target attribute has been ignored for overload resolution}}
  targetdep_t __attribute__((host)) decl_ret_late_host(void);
  targetdep_t __attribute__((host,device)) decl_ret_late_hostdevice(void); // expected-warning {{target attribute has been ignored for overload resolution}}
};

void tests(void) {
  MethodTests mt;

  ASSERT_HAS_TYPE(mt.ret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(mt.ret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(mt.ret_early_hostdevice({}), CurrentTargetTy)

  ASSERT_HAS_TYPE(mt.autoret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(mt.autoret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(mt.autoret_early_hostdevice({}), CurrentTargetTy)

  // The target attribute is too late to be considered:
  ASSERT_HAS_TYPE(mt.ret_late_device({}), HostTy)
  ASSERT_HAS_TYPE(mt.ret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(mt.ret_late_hostdevice({}), HostTy)

  ASSERT_HAS_TYPE(mt.decl_ret_early_device(), DeviceTy)
  ASSERT_HAS_TYPE(mt.decl_ret_early_host(), HostTy)
  ASSERT_HAS_TYPE(mt.decl_ret_early_hostdevice(), CurrentTargetTy)

  // The target attribute is too late to be considered:
  ASSERT_HAS_TYPE(mt.decl_ret_late_device(), HostTy)
  ASSERT_HAS_TYPE(mt.decl_ret_late_host(), HostTy)
  ASSERT_HAS_TYPE(mt.decl_ret_late_hostdevice(), HostTy)

  TemplateMethodTests<42> tmt;
  ASSERT_HAS_TYPE(tmt.ret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(tmt.ret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(tmt.ret_early_hostdevice({}), CurrentTargetTy)

  ASSERT_HAS_TYPE(tmt.autoret_early_device({}), DeviceTy)
  ASSERT_HAS_TYPE(tmt.autoret_early_host({}), HostTy)
  ASSERT_HAS_TYPE(tmt.autoret_early_hostdevice({}), CurrentTargetTy)

  ASSERT_HAS_TYPE(tmt.ret_late_device({}), HostTy)
  ASSERT_HAS_TYPE(tmt.ret_late_host({}), HostTy)
  ASSERT_HAS_TYPE(tmt.ret_late_hostdevice({}), HostTy)

  ASSERT_HAS_TYPE(tmt.decl_ret_early_device(), DeviceTy)
  ASSERT_HAS_TYPE(tmt.decl_ret_early_host(), HostTy)
  ASSERT_HAS_TYPE(tmt.decl_ret_early_hostdevice(), CurrentTargetTy)

  ASSERT_HAS_TYPE(tmt.decl_ret_late_device(), HostTy)
  ASSERT_HAS_TYPE(tmt.decl_ret_late_host(), HostTy)
  ASSERT_HAS_TYPE(tmt.decl_ret_late_hostdevice(), HostTy)
}


// global variables:
__attribute__((device)) targetdep_t var_early_device = {};
ASSERT_HAS_TYPE(var_early_device, DeviceTy)

targetdep_t var_early_host = {};
ASSERT_HAS_TYPE(var_early_host, HostTy)

targetdep_t __attribute__((device)) var_late_device = {}; // expected-warning {{target attribute has been ignored for overload resolution}}
ASSERT_HAS_TYPE(var_late_device, HostTy)


// Tests for the overload candidate ordering compared to templates:

enum Candidate {
  TEMPLATE,
  HOST,
  DEVICE,
  HOSTDEVICE,
};

// (1.) If the overloaded functions are constexpr

// (1.a) Prefer fitting overloads.
template <typename T> constexpr Candidate ce_template_vs_H_D_functions(T arg) { return TEMPLATE; }
__attribute__((device)) constexpr Candidate ce_template_vs_H_D_functions(float arg) { return DEVICE; }
__attribute__((host)) constexpr Candidate ce_template_vs_H_D_functions(float arg) { return HOST; }

__attribute__((device)) check<ce_template_vs_H_D_functions(1.0f) == DEVICE>::type
test_ce_template_vs_H_D_functions_for_device() {
  return TrueTy();
}

__attribute__((host)) check<ce_template_vs_H_D_functions(1.0f) == HOST>::type
test_ce_template_vs_H_D_functions_for_host() {
  return TrueTy();
}

__attribute__((host,device)) check<ce_template_vs_H_D_functions(1.0f) == CurrentTarget>::type
test_ce_template_vs_H_D_functions_for_hd() {
  return TrueTy();
}


// (1.b) Always prefer an HD candidate over a template candidate.
template <typename T> constexpr Candidate ce_template_vs_HD_function(T arg) { return TEMPLATE; }
__attribute__((host, device)) constexpr Candidate ce_template_vs_HD_function(float arg) { return HOSTDEVICE; }

__attribute__((device)) check<ce_template_vs_HD_function(1.0f) == HOSTDEVICE>::type
test_ce_template_vs_HD_function_for_device() {
  return TrueTy();
}

__attribute__((host)) check<ce_template_vs_HD_function(1.0f) == HOSTDEVICE>::type
test_ce_template_vs_HD_function_for_host() {
  return TrueTy();
}

__attribute__((host,device)) check<ce_template_vs_HD_function(1.0f) == HOSTDEVICE>::type
test_ce_template_vs_HD_function_for_hd() {
  return TrueTy();
}


// (1.c) Even wrong-sided calls are okay if the called function is constexpr, so
// prefer the device overload over the template.
template <typename T> constexpr Candidate ce_template_vs_D_function(T arg) { return TEMPLATE; }
__attribute__((device)) constexpr Candidate ce_template_vs_D_function(float arg) { return DEVICE; }

__attribute__((host)) check<ce_template_vs_D_function(1.0f) == DEVICE>::type
test_ce_template_vs_D_function_for_host() {
  return TrueTy();
}

__attribute__((device)) check<ce_template_vs_D_function(1.0f) == DEVICE>::type
test_ce_template_vs_D_function_for_device() {
  return TrueTy();
}

__attribute__((host,device)) check<ce_template_vs_D_function(1.0f) == DEVICE>::type
test_ce_template_vs_D_function_for_hd() {
  return TrueTy();
}


// (2.) If the overloaded functions are NOT constexpr

// (2.a) Prefer fitting overloads.
template <typename T> TemplateTy template_vs_H_D_functions(T arg) { return {}; }
__attribute__((device)) DeviceTy template_vs_H_D_functions(float arg) { return {}; }
__attribute__((host)) HostTy template_vs_H_D_functions(float arg) { return {}; }

__attribute__((device)) check<is_same<decltype(template_vs_H_D_functions(1.0f)), DeviceTy>::value>::type
test_template_vs_H_D_functions_for_device() {
  return TrueTy{};
}

__attribute__((host)) check<is_same<decltype(template_vs_H_D_functions(1.0f)), HostTy>::value>::type
test_template_vs_H_D_functions_for_host() {
  return TrueTy{};
}

__attribute__((host,device)) check<is_same<decltype(template_vs_H_D_functions(1.0f)), CurrentTargetTy>::value>::type
test_template_vs_H_D_functions_for_hd() {
  return TrueTy{};
}

// (2.b) Always prefer an HD candidate over a template candidate.
template <typename T> TemplateTy template_vs_HD_function(T arg) { return {}; }
__attribute__((host,device)) HostDeviceTy template_vs_HD_function(float arg) { return {}; }

__attribute__((device)) check<is_same<decltype(template_vs_HD_function(1.0f)), HostDeviceTy>::value>::type
test_template_vs_HD_function_for_device() {
  return TrueTy{};
}

__attribute__((host)) check<is_same<decltype(template_vs_HD_function(1.0f)), HostDeviceTy>::value>::type
test_template_vs_HD_function_for_host() {
  return TrueTy{};
}

__attribute__((host,device)) check<is_same<decltype(template_vs_HD_function(1.0f)), HostDeviceTy>::value>::type
test_template_vs_HD_function_for_hd() {
  return TrueTy{};
}


// (2.c) For non-constexpr functions, prefer a sameside or native template
// function over a wrongside non-template function:
template <typename T> TemplateTy template_vs_D_function(T arg) { return {}; }
__attribute__((device)) DeviceTy template_vs_D_function(float arg) { return {}; }

__attribute__((host,device)) check<is_same<decltype(template_vs_D_function(1.0f)), TemplateIfHostTy>::value>::type
test_template_vs_D_function_for_hd() {
  return TrueTy{};
}

__attribute__((device)) check<is_same<decltype(template_vs_D_function(1.0f)), DeviceTy>::value>::type
test_template_vs_D_function_for_device() {
  return TrueTy{};
}

__attribute__((host)) check<is_same<decltype(template_vs_D_function(1.0f)), TemplateTy>::value>::type
test_template_vs_D_function_for_host() {
  return TrueTy{};
}


// If only a wrongside function is available, it is selected.
__attribute__((device)) DeviceTy only_D_function(float arg) { return {}; }

__attribute__((host)) check<is_same<decltype(only_D_function(1.0f)), DeviceTy>::value>::type
test_only_D_function_for_host() {
  return TrueTy{};
}

// Default arguments for template parameters occur before the target attribute,
// so we can't identify the "right" overload for them.
template <typename T = targetdep_t>
__attribute__((device)) // expected-warning {{target attribute has been ignored for overload resolution}}
T use_in_template_default_arg(void) {
  return HostTy{};
}

__attribute__((device))
void test_use_in_template(void) {
  use_in_template_default_arg<>();
}
