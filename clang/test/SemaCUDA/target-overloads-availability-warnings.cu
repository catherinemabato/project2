
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -fsyntax-only -verify=expected,onhost %s
// RUN: %clang_cc1 -triple nvptx64-nvidia-cuda -fsyntax-only -fcuda-is-device -verify=expected,ondevice %s

template <bool C, class T = void> struct my_enable_if {};

template <class T> struct my_enable_if<true, T> {
  typedef T type;
};

__attribute__((host, device)) void use(int x);

// For 'OverloadFunHostDepr', the host overload is deprecated, the device overload is not.
__attribute__((device)) constexpr int OverloadFunHostDepr(void) { return 1; }
__attribute__((host, deprecated("Host variant"))) constexpr int OverloadFunHostDepr(void) { return 1; } // expected-note 0+ {{has been explicitly marked deprecated here}}


// For 'OverloadFunDeviceDepr', the device overload is deprecated, the host overload is not.
__attribute__((device, deprecated("Device variant"))) constexpr int OverloadFunDeviceDepr(void) { return 1; } // expected-note 0+ {{has been explicitly marked deprecated here}}
__attribute__((host)) constexpr int OverloadFunDeviceDepr(void) { return 1; }


// For 'TemplateOverloadFun', the host overload is deprecated, the device overload is not.
template<typename T>
__attribute__((device)) constexpr T TemplateOverloadFun(void) { return 1; }

template<typename T>
__attribute__((host, deprecated("Host variant"))) constexpr T TemplateOverloadFun(void) { return 1; } // expected-note 0+ {{has been explicitly marked deprecated here}}


// There is only a device overload, and it is deprecated.
__attribute__((device, deprecated)) constexpr int // expected-note 0+ {{has been explicitly marked deprecated here}}
DeviceOnlyFunDeprecated(void) { return 1; }

// There is only a host overload, and it is deprecated.
__attribute__((host, deprecated)) constexpr int // expected-note 0+ {{has been explicitly marked deprecated here}}
HostOnlyFunDeprecated(void) { return 1; }

class FunSelector {
public:
  // This should use the non-deprecated device overload.
  template<int X> __attribute__((device))
  auto devicefun(void) -> typename my_enable_if<(X == OverloadFunHostDepr()), int>::type {
    return 1;
  }

  // This should use the non-deprecated device overload.
  template<int X> __attribute__((device))
  auto devicefun(void) -> typename my_enable_if<(X != OverloadFunHostDepr()), int>::type {
      return 0;
  }

  // This should use the deprecated device overload.
  template<int X> __attribute__((device))
  auto devicefun_wrong(void) -> typename my_enable_if<(X == OverloadFunDeviceDepr()), int>::type { // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
    return 1;
  }

  // This should use the deprecated device overload.
  template<int X> __attribute__((device))
  auto devicefun_wrong(void) -> typename my_enable_if<(X != OverloadFunDeviceDepr()), int>::type { // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
      return 0;
  }

  // This should use the non-deprecated host overload.
  template<int X> __attribute__((host))
  auto hostfun(void) -> typename my_enable_if<(X == OverloadFunDeviceDepr()), int>::type {
    return 1;
  }

  // This should use the non-deprecated host overload.
  template<int X> __attribute__((host))
  auto hostfun(void) -> typename my_enable_if<(X != OverloadFunDeviceDepr()), int>::type {
      return 0;
  }

  // This should use the deprecated host overload.
  template<int X> __attribute__((host))
  auto hostfun_wrong(void) -> typename my_enable_if<(X == OverloadFunHostDepr()), int>::type { // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
    return 1;
  }

  // This should use the deprecated host overload.
  template<int X> __attribute__((host))
  auto hostfun_wrong(void) -> typename my_enable_if<(X != OverloadFunHostDepr()), int>::type { // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
      return 0;
  }
};


// These should not be diagnosed since the device overload of
// OverloadFunHostDepr is not deprecated:
__attribute__((device)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type
DeviceUserOverloadFunHostDepr1(void) { return 2; }

__attribute__((device)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type constexpr
DeviceUserOverloadFunHostDeprConstexpr(void) { return 2; }


// Analogously for OverloadFunDeviceDepr:
__attribute__((host)) my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type
DeviceUserOverloadFunDeviceDepr1(void) { return 2; }

my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type __attribute__((host))
DeviceUserOverloadFunDeviceDepr2(void) { return 2; }

__attribute__((host)) my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type constexpr
DeviceUserOverloadFunDeviceDeprConstexpr(void) { return 2; }


// Actual uses of the deprecated overloads should be diagnosed:
__attribute__((host, device)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type // onhost-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
HostDeviceUserOverloadFunHostDepr(void) { return 3; }

__attribute__((host)) my_enable_if<(OverloadFunHostDepr() > 0), int>::type constexpr // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
HostUserOverloadFunHostDeprConstexpr(void) { return 3; }

__attribute__((device)) my_enable_if<(OverloadFunDeviceDepr() > 0), int>::type constexpr // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
HostUserOverloadFunDeviceDeprConstexpr(void) { return 3; }


// Making the offending decl a template shouldn't change anything:
__attribute__((host)) my_enable_if<(TemplateOverloadFun<int>() > 0), int>::type // expected-warning {{'TemplateOverloadFun<int>' is deprecated: Host variant}}
HostUserTemplateOverloadFun(void) { return 3; }

__attribute__((device)) my_enable_if<(TemplateOverloadFun<int>() > 0), int>::type
DeviceUserTemplateOverloadFun(void) { return 3; }


__attribute__((device, deprecated)) constexpr int DeviceVarConstDepr = 1; // expected-note 0+ {{has been explicitly marked deprecated here}}

// Diagnostics for uses in function bodies should work as expected:
__attribute__((host)) void HostUser(void) {
  use(DeviceVarConstDepr); // expected-warning {{'DeviceVarConstDepr' is deprecated}}
  use(HostOnlyFunDeprecated()); // expected-warning {{'HostOnlyFunDeprecated' is deprecated}}
  use(OverloadFunHostDepr()); // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
  use(TemplateOverloadFun<int>()); // expected-warning {{'TemplateOverloadFun<int>' is deprecated: Host variant}}

  use(OverloadFunDeviceDepr());
}

__attribute__((device)) void DeviceUser(void) {
  use(DeviceVarConstDepr); // expected-warning {{'DeviceVarConstDepr' is deprecated}}
  use(DeviceOnlyFunDeprecated()); // expected-warning {{'DeviceOnlyFunDeprecated' is deprecated}}
  use(OverloadFunDeviceDepr()); // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}

  use(OverloadFunHostDepr());
  use(TemplateOverloadFun<int>());
}


// Template functions outside of classes:

// This should use the non-deprecated device overload.
template<int X> __attribute__((device))
auto devicefun(void) -> typename my_enable_if<(X == OverloadFunHostDepr()), int>::type {
  return 1;
}

// This should use the non-deprecated device overload.
template<int X> __attribute__((device))
auto devicefun(void) -> typename my_enable_if<(X != OverloadFunHostDepr()), int>::type {
    return 0;
}

// This should use the deprecated device overload.
template<int X> __attribute__((device))
auto devicefun_wrong(void) -> typename my_enable_if<(X == OverloadFunDeviceDepr()), int>::type { // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
  return 1;
}

// This should use the deprecated device overload.
template<int X> __attribute__((device))
auto devicefun_wrong(void) -> typename my_enable_if<(X != OverloadFunDeviceDepr()), int>::type { // expected-warning {{'OverloadFunDeviceDepr' is deprecated: Device variant}}
    return 0;
}

// This should use the non-deprecated host overload.
template<int X> __attribute__((host))
auto hostfun(void) -> typename my_enable_if<(X == OverloadFunDeviceDepr()), int>::type {
  return 1;
}

// This should use the non-deprecated host overload.
template<int X> __attribute__((host))
auto hostfun(void) -> typename my_enable_if<(X != OverloadFunDeviceDepr()), int>::type {
    return 0;
}

// This should use the deprecated host overload.
template<int X> __attribute__((host))
auto hostfun_wrong(void) -> typename my_enable_if<(X == OverloadFunHostDepr()), int>::type { // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
  return 1;
}

// This should use the deprecated host overload.
template<int X> __attribute__((host))
auto hostfun_wrong(void) -> typename my_enable_if<(X != OverloadFunHostDepr()), int>::type { // expected-warning {{'OverloadFunHostDepr' is deprecated: Host variant}}
    return 0;
}
